/*
 *  SPDX-FileCopyrightText: Copyright 2021, Siavash Ameli <sameli@berkeley.edu>
 *  SPDX-License-Identifier: BSD-3-Clause
 *  SPDX-FileType: SOURCE
 *
 *  This program is free software: you can redistribute it and/or modify it
 *  under the terms of the license found in the LICENSE.txt file in the root
 *  directory of this source tree.
 */


// =======
// Headers
// =======

#include "./cu_linear_operator.h"
#include <cstddef>  // NULL
#include <cassert>  // assert
#include <cstdlib>  // abort
#include <iostream>


// =============
// constructor 1
// =============

template <typename DataType>
cuLinearOperator<DataType>::cuLinearOperator():
    copied_host_to_device(false),
    cublas_handle(NULL),
    cusparse_handle(NULL)
{
    // Check any gpu device exists
    this->query_gpu_devices();

    // Regardless of using dense (cublas) or sparse (cusparse) matrices, the
    // cublas handle should be initialized, since it is needed for the methods
    // in cuVectorOperations
    this->initialize_cublas_handle();
}


// =============
// constructor 2
// =============

/// \brief  Constructor with setting \c num_rows and \c num_columns.
///
/// \note   For the classed that are virtually derived (virtual inheritance)
///         from this class, this constructor will never be called. Rather, the
///         default constructor is called by the most derived class. Thus, set
///         the member data directly instead of below.

template <typename DataType>
cuLinearOperator<DataType>::cuLinearOperator(
        const LongIndexType num_rows_,
        const LongIndexType num_columns_):

    // Base class constructor
    cLinearOperator<DataType>(num_rows_, num_columns_),

    // Initializer list
    copied_host_to_device(false),
    cublas_handle(NULL),
    cusparse_handle(NULL)
{
    // Check any gpu device exists
    this->query_gpu_devices();

    // Regardless of using dense (cublas) or sparse (cusparse) matrices, the
    // cublas handle should be initialized, since it is needed for the methods
    // in cuVectorOperations
    this->initialize_cublas_handle();
}


// ==========
// destructor
// ==========

template <typename DataType>
cuLinearOperator<DataType>::~cuLinearOperator()
{
    if (this->cublas_handle != NULL)
    {
        hipblasStatus_t status = hipblasDestroy(this->cublas_handle);
        assert(status == HIPBLAS_STATUS_SUCCESS);
        this->cublas_handle = NULL;
    }

    if (this->cusparse_handle != NULL)
    {
        hipsparseStatus_t status = hipsparseDestroy(this->cusparse_handle);
        assert(status == HIPSPARSE_STATUS_SUCCESS);
        this->cusparse_handle = NULL;
    }
}


// =================
// get cublas handle
// =================

/// \brief   This function returns a reference to the \c hipblasHandle_t
///          object. The object will be created, if it is not created already.
///
/// \details The \c cublasHandle is needed for the client code (slq method) for
///          vector operations on GPU. However, in this class, the
///          \c hipblasHandle_t might not be needed by it self if the derived
///          class is a sparse matrix, becase the sparse matrix needs only
///          \c hipsparseHandle_t. In case if the \c hipblasHandle_t is not
///          created, it will be created for the purpose of the client codes.
///
/// \return  A void pointer to the hipblasHandle_t instance.

template <typename DataType>
hipblasHandle_t cuLinearOperator<DataType>::get_cublas_handle() const
{
    return this->cublas_handle;
}


// ========================
// initialize cublas handle
// ========================

/// \brief Creates a \c hipblasHandle_t object, if not created already.
///

template <typename DataType>
void cuLinearOperator<DataType>::initialize_cublas_handle()
{
    if (this->cublas_handle == NULL)
    {
        hipblasStatus_t status = hipblasCreate(&this->cublas_handle);
        assert(status == HIPBLAS_STATUS_SUCCESS);
    }
}


// ==========================
// initialize cusparse handle
// ==========================

/// \brief Creates a \c hipsparseHandle_t object, if not created already.
///

template <typename DataType>
void cuLinearOperator<DataType>::initialize_cusparse_handle()
{
    if (this->cusparse_handle == NULL)
    {
        hipsparseStatus_t status = hipsparseCreate(&this->cusparse_handle);
        assert(status == HIPSPARSE_STATUS_SUCCESS);
    }
}


// =================
// query gpu devices
// =================

/// \brief Before any numerical computation, this method chechs if any gpu
///        device is available on the machine, or notifies the user if nothing
///        was found.
///
/// \return Number of gpu available devices.

template <typename DataType>
int cuLinearOperator<DataType>::query_gpu_devices() const
{
    int device_count = 0;
    hipError_t error = hipGetDeviceCount(&device_count);

    // Error code 38 means no cuda-capable device was detected.
    if (error != hipSuccess)
    {
        std::cerr << "ERROR: No cuda-capable GPU device was detected on " \
                  << "this machine. If a cuda-capable GPU device exists, " \
                  << "install its cuda driver. Alternatively, set " \
                  << "'gpu=False' to use cpu instead." \
                  << std::endl;
        abort();
    }

    return device_count;
}


// ===============================
// Explicit template instantiation
// ===============================

template class cuLinearOperator<float>;
template class cuLinearOperator<double>;
